#include "hip/hip_runtime.h"
// Copyright (c) 2018-2020, Michael P. Howard
// This file is part of the azplugins project, released under the Modified BSD License.

// Maintainer: astatt

/*!
 * \file DynamicBondUpdaterGPU.cu
 * \brief Definition of kernel drivers and kernels for DynamicBondUpdaterGPU
 */

#include "hoomd/HOOMDMath.h"
#include "DynamicBondUpdaterGPU.cuh"
//#include <thrust/sort.h>
#include <thrust/device_vector.h>
// todo: should azplugins have its own "extern"?
#include "hoomd/extern/neighbor/neighbor/LBVH.cuh"
#include "hoomd/extern/neighbor/neighbor/LBVHTraverser.cuh"
//#include "hoomd/extern/cub/hipcub/hipcub.hpp"


namespace azplugins
{

//todo: migrate to separate file/class.
//this sorts according distance, then first tag, then second tag
struct SortBondsGPU{
  __host__ __device__ bool operator()(const Scalar3 &i, const Scalar3 &j)
    {
      const Scalar r_sq_1 = i.z;
      const Scalar r_sq_2 = j.z;
      if (r_sq_1==r_sq_2)
      {
        const unsigned int tag_11 = __scalar_as_int(i.x);
        const unsigned int tag_21 = __scalar_as_int(j.x);
        if (tag_11==tag_21)
        {
        const unsigned int tag_12 = __scalar_as_int(i.y);
        const unsigned int tag_22 = __scalar_as_int(j.y);
        return tag_22>tag_12;
        }
        else
        {
          return tag_21>tag_11;
        }
      }
      else
      {
        return r_sq_2>r_sq_1;
      }
    }

};

// returns true if given possible bond is zero, e.g. (0,0,0.0)
// possible bonds are ordered, such that tag_a < tag_b in (tag_a,tag_b,rsq)
// meaning we only need to check tag_b == 0
struct isZeroBondGPU{
  __host__ __device__ bool operator()(const Scalar3 &i)
    {
      const unsigned int tag_1 = __scalar_as_int(i.y);
      return !(bool)tag_1;
    }
};

struct CompareBondsGPU{
  __host__ __device__ bool operator()(const Scalar3 &i, const Scalar3 &j)
    {
      const unsigned int tag_11 = __scalar_as_int(i.x);
      const unsigned int tag_12 = __scalar_as_int(i.y);
      const unsigned int tag_21 = __scalar_as_int(j.x);
      const unsigned int tag_22 = __scalar_as_int(j.y);

      if ((tag_11==tag_21 && tag_12==tag_22))   // should work because pairs are ordered
      {
        return true;
      }
      else
      {
        return false;
      }
    }
  };

namespace gpu
{

//! Number of elements of the exclusion list to process in each batch
const unsigned int FILTER_BATCH_SIZE = 4;

namespace kernel
{

__global__ void copy_nlist_possible_bonds(Scalar3 *d_all_possible_bonds,
                                  const Scalar4 *d_postype,
                                  const unsigned int * d_tag,
                                  const unsigned int * d_sorted_indexes,
                                  const unsigned int * d_n_neigh,
                                  const unsigned int * d_nlist,
                                  const BoxDim box,
                                  const unsigned int max_bonds,
                                  const Scalar r_cut,
                                  const bool groups_identical,
                                  const unsigned int N)
        {

        // one thread per particle in group_1
        const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

        if (idx >= N)
            return;

        // idx = group index , pidx = actual particle index
        const unsigned int pidx_i = d_sorted_indexes[idx];
        unsigned int n_curr_bond = 0;
        const Scalar r_cutsq = r_cut*r_cut;

        // get all information for this particle
        Scalar4 postype_i = d_postype[pidx_i];
        const unsigned int tag_i = d_tag[pidx_i];
        const unsigned int n_neigh = d_n_neigh[idx];

        // loop over all neighbors of this particle
        for (unsigned int j=0; j<n_neigh;++j)
          {
              // get index of neighbor from neigh_list
              const unsigned int pidx_j = d_nlist[idx*max_bonds + j];
              Scalar4 postype_j = d_postype[pidx_j];
              const unsigned int tag_j = d_tag[pidx_j];

              Scalar3 drij = make_scalar3(postype_j.x,postype_j.y,postype_j.z)
                           - make_scalar3(postype_i.x,postype_i.y,postype_i.z);

             // apply periodic boundary conditions (FLOPS: 12)
              drij = box.minImage(drij);

              // same as on the cpu, just not during the tree traversal
               Scalar dr_sq = dot(drij,drij);

               if (dr_sq < r_cutsq)
                   {
                   if (n_curr_bond < max_bonds)
                      {
                      Scalar3 d;
                      if (groups_identical)
                        {
                        // sort the two tags in this possible bond pair if groups are identical
                        const unsigned int tag_a = tag_j>tag_i ? tag_i : tag_j;
                        const unsigned int tag_b = tag_j>tag_i ? tag_j : tag_i;
                        d = make_scalar3(__int_as_scalar(tag_a),__int_as_scalar(tag_b),dr_sq);
                        }
                      else
                        {
                        d = make_scalar3(__int_as_scalar(tag_i),__int_as_scalar(tag_j),dr_sq);
                        }
                      d_all_possible_bonds[idx*max_bonds + n_curr_bond] = d;
                      }
                    ++n_curr_bond;
                  }
          }
        }

/*! \param d_all_possible_bonds all possible bonds list
    \param d_n_existing_bonds Number of existing for each particle
    \param d_existing_bonds_list List of exitsting for each particle
    \param exli Indexer for indexing into d_existing_bonds_list
    \param size Length of d_all_possible_bonds
    \param ex_start Start filtering  d_all_possible_bonds from existing bond number \a ex_start

    the kernel filter_existing_bonds() processes the all possible bonds list \a d_nlist and removes any entries that already exist. To allow
    for an arbitrary large number of existing bonds, these are processed in batch sizes of FILTER_BATCH_SIZE. The kernel
    must be called multiple times in order to fully remove all exclusions from the nlist.

    \note The driver filter_existing_bonds properly makes as many calls as are necessary, it only needs to be called once.

    \b Implementation

    One thread is run for each particle. Existing bonds \a ex_start, \a ex_start + 1, ... are loaded in for that particle
    (or the thread returns if there are no exclusions past that point). The thread then loops over the neighbor list,
    comparing each entry to the list of exclusions. If the entry is not excluded, it is written back out. \a d_n_neigh
    is updated to reflect the current number of particles in the list at the end of the kernel call.
*/
__global__ void filter_existing_bonds(Scalar3 *d_all_possible_bonds,
                                      const unsigned int *d_n_existing_bonds,
                                      const unsigned int *d_existing_bonds_list,
                                      const Index2D exli,
                                      const unsigned int size,
                                      const unsigned int ex_start)
  {
  // compute the bond index this thread operates on
  const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

  // quit now if this thread is processing past the end of the list of all possible bonds
  if (idx >= size)
      return;

  Scalar3 current_bond = d_all_possible_bonds[idx];
  unsigned int tag_1 = __scalar_as_int(current_bond.x);
  unsigned int tag_2 = __scalar_as_int(current_bond.y);

  if(tag_1==0 && tag_2==0)
      return;

  //const unsigned int n_neigh = d_n_neigh[idx];
  const unsigned int n_ex = d_n_existing_bonds[tag_1];

  // quit now if the ex_start flag is past the end of n_ex
  if (ex_start >= n_ex)
      return;

  // count the number of existing bonds to process in this thread
  const unsigned int n_ex_process = n_ex - ex_start;

  // load the existing bond list into "local" memory - fully unrolled loops should dump this into registers
  unsigned int l_existing_bonds_list[FILTER_BATCH_SIZE];
  #pragma unroll
  for (unsigned int cur_ex_idx = 0; cur_ex_idx < FILTER_BATCH_SIZE; cur_ex_idx++)
      {
      if (cur_ex_idx < n_ex_process)
      {
          l_existing_bonds_list[cur_ex_idx] = d_existing_bonds_list[exli(tag_1, cur_ex_idx + ex_start)];
        }
      else
      {
          l_existing_bonds_list[cur_ex_idx] = 0xffffffff;
        }
      }

      // test if excluded
      bool excluded = false;
      #pragma unroll
      for (unsigned int cur_ex_idx = 0; cur_ex_idx < FILTER_BATCH_SIZE; cur_ex_idx++)
          {
          if (tag_2 == l_existing_bonds_list[cur_ex_idx])
              excluded = true;
          }
      // if it is excluded, overwrite that entry with (0,0,0).
      if (excluded)
          {
            d_all_possible_bonds[idx] = make_scalar3(__int_as_scalar(0),__int_as_scalar(0),0.0);
          }
  }

} // end namespace kernel


hipError_t remove_zeros_and_sort_possible_bond_array(Scalar3 *d_all_possible_bonds,
                                                      const unsigned int size,
                                                      int *d_max_non_zero_bonds)
    {
    if (size == 0) return hipSuccess;
    // wrapper for pointer needed for thrust
    HOOMD_THRUST::device_ptr<Scalar3> d_all_possible_bonds_wrap(d_all_possible_bonds);

    isZeroBondGPU zero;
    HOOMD_THRUST::device_ptr<Scalar3> last0 = HOOMD_THRUST::remove_if(d_all_possible_bonds_wrap,d_all_possible_bonds_wrap + size, zero);
    unsigned int l0 = HOOMD_THRUST::distance(d_all_possible_bonds_wrap, last0);

    // sort remainder by distance, should make all identical bonds consequtive
    SortBondsGPU sort;
    HOOMD_THRUST::sort(d_all_possible_bonds_wrap,d_all_possible_bonds_wrap+l0, sort);

    // thrust::unique only removes identical consequtive elements, so sort above is needed.
    CompareBondsGPU comp;
    HOOMD_THRUST::device_ptr<Scalar3> last1 = HOOMD_THRUST::unique(d_all_possible_bonds_wrap, d_all_possible_bonds_wrap + l0,comp);
    unsigned int l1 = HOOMD_THRUST::distance(d_all_possible_bonds_wrap, last1);

    *d_max_non_zero_bonds=l1;

    return hipSuccess;
    }


hipError_t filter_existing_bonds(Scalar3 *d_all_possible_bonds,
                             unsigned int *d_n_existing_bonds,
                             const unsigned int *d_existing_bonds_list,
                             const Index2D& exli,
                             const unsigned int size,
                             const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))kernel::filter_existing_bonds);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);

    // determine parameters for kernel launch
    int n_blocks = size/run_block_size + 1;

    // split the processing of the full exclusion list up into a number of batches
    unsigned int n_batches = (unsigned int)ceil(double(exli.getH())/double(FILTER_BATCH_SIZE));
    unsigned int ex_start = 0;
    for (unsigned int batch = 0; batch < n_batches; batch++)
        {
        kernel::filter_existing_bonds<<<n_blocks, run_block_size>>>(d_all_possible_bonds,
                                                              d_n_existing_bonds,
                                                              d_existing_bonds_list,
                                                              exli,
                                                              size,
                                                              ex_start);

        ex_start += FILTER_BATCH_SIZE;
        }

    return hipSuccess;
    }


hipError_t copy_possible_bonds(Scalar3 *d_all_possible_bonds,
                          const Scalar4 *d_postype,
                          const unsigned int *d_tag,
                          const unsigned int *d_sorted_indexes,
                          const unsigned int *d_n_neigh,
                          const unsigned int *d_nlist,
                          const BoxDim box,
                          const unsigned int max_bonds,
                          const Scalar r_cut,
                          const bool groups_identical,
                          const unsigned int N,
                          const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;

    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))kernel::copy_nlist_possible_bonds);
        max_block_size = attr.maxThreadsPerBlock;
        }
    unsigned int run_block_size = min(block_size, max_block_size);

    kernel::copy_nlist_possible_bonds<<<N/run_block_size + 1, run_block_size>>>(d_all_possible_bonds,
                                                                         d_postype,
                                                                         d_tag,
                                                                         d_sorted_indexes,
                                                                         d_n_neigh,
                                                                         d_nlist,
                                                                         box,
                                                                         max_bonds,
                                                                         r_cut,
                                                                         groups_identical,
                                                                         N);
    return hipSuccess;
    }

} // end namespace gpu
} // end namespace azplugins


// explicit templates for neighbor::LBVH with PointMapInsertOp
template void neighbor::gpu::lbvh_gen_codes(unsigned int *, unsigned int *, const azplugins::gpu::PointMapInsertOp&,
const Scalar3, const Scalar3, const unsigned int, const unsigned int, hipStream_t);
template void neighbor::gpu::lbvh_bubble_aabbs(const neighbor::gpu::LBVHData, const azplugins::gpu::PointMapInsertOp&,
unsigned int *, const unsigned int, const unsigned int, hipStream_t);
template void neighbor::gpu::lbvh_one_primitive(const neighbor::gpu::LBVHData, const azplugins::gpu::PointMapInsertOp&, hipStream_t);
template void neighbor::gpu::lbvh_traverse_ropes(azplugins::gpu::NeighborListOp&, const neighbor::gpu::LBVHCompressedData&,
const azplugins::gpu::ParticleQueryOp&, const Scalar3 *, unsigned int, unsigned int, hipStream_t);
