#include "hip/hip_runtime.h"
// Copyright (c) 2018-2020, Michael P. Howard
// This file is part of the azplugins project, released under the Modified BSD License.

// Maintainer: mphoward

/*!
 * \file TwoStepSLLODLangevinFlowGPU.cuh
 * \brief Definition of kernel drivers and kernels for TwoStepSLLODLangevinFlowGPU
 */

#include "TwoStepSLLODLangevinFlowGPU.cuh"


namespace azplugins
{
namespace gpu
{
namespace kernel
{
__global__ void langevin_sllod_step1(Scalar4 *d_pos,
                                    int3 *d_image,
                                    Scalar4 *d_vel,
                                    const Scalar3 *d_accel,
                                    const unsigned int *d_group,
                                    const BoxDim box,
                                    const unsigned int N,
                                    const Scalar dt,
                                    const Scalar shear_rate,
                                    const bool flipped,
                                    const Scalar boundary_shear_velocity)
    {
    const unsigned int grp_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (grp_idx >= N) return;

    const unsigned int idx = d_group[grp_idx];

    // position
    const Scalar4 postype = d_pos[idx];
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    unsigned int type = __scalar_as_int(postype.w);

    // velocity
    const Scalar4 velmass = d_vel[idx];
    Scalar3 vel = make_scalar3(velmass.x, velmass.y, velmass.z);
    Scalar mass = velmass.w;

    // acceleration
    const Scalar3 accel = d_accel[idx];


    // remove flow field
    vel.x -= shear_rate*pos.y;

    // apply sllod velocity correction
    vel.x -= Scalar(0.5)*shear_rate*vel.y*dt;

    // add flow field
    vel.x += shear_rate*pos.y;

    // update velocity
    vel += Scalar(0.5)*accel*dt;

    // update position
    pos += dt * vel;

    // if box deformation caused a flip, wrap positions back into box
    if (flipped){
         d_image[idx].x += d_image[idx].y;
        //pos.x *= -1;
    }

    // read in the image flags
    int3 image = d_image[idx];

    // time to fix the periodic boundary conditions
    box.wrap(pos, image);

    // Periodic boundary correction to velocity:
    // if particle leaves from (+/-) y boundary it gets (-/+) velocity at boundary
    // note carefully that pair potentials dependent on differences in
    // velocities (e.g. DPD) are not yet explicitly supported.

    if ((image.y-d_image[idx].y)==1) // crossed pbc in +y, image increased by 1
    {
      vel.x -= boundary_shear_velocity;
    }
    else if ((image.y-d_image[idx].y)==-1) // crossed pbc in -y, image decreased by 1
    {
      vel.x += boundary_shear_velocity;
    }

    // save results
    d_pos[idx] = make_scalar4(pos.x, pos.y, pos.z, __int_as_scalar(type));
    d_vel[idx] = make_scalar4(vel.x, vel.y, vel.z, mass);
    d_image[idx] = image;
    }


__global__ void langevin_sllod_step2(Scalar4 *d_vel,
                                    Scalar3 *d_accel,
                                    const Scalar4 *d_pos,
                                    const Scalar4 *d_net_force,
                                    const unsigned int *d_tag,
                                    const unsigned int *d_group,
                                    const Scalar *d_diameter,
                                    const Scalar lambda,
                                    const Scalar *d_gamma,
                                    const unsigned int ntypes,
                                    const unsigned int N,
                                    const Scalar dt,
                                    const Scalar T,
                                    const unsigned int timestep,
                                    const unsigned int seed,
                                    bool noiseless,
                                    bool use_lambda,
                                    const Scalar shear_rate)
    {
    // optionally cache gamma into shared memory
    extern __shared__ Scalar s_gammas[];
    if (!use_lambda)
        {
        for (int cur_offset = 0; cur_offset < ntypes; cur_offset += blockDim.x)
            {
            if (cur_offset + threadIdx.x < ntypes)
                s_gammas[cur_offset + threadIdx.x] = d_gamma[cur_offset + threadIdx.x];
            }
        __syncthreads();
        }

    // one thread per particle in group
    const unsigned int grp_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (grp_idx >= N) return;
    const unsigned int idx = d_group[grp_idx];

    // get the friction coefficient
    const Scalar4 postype = d_pos[idx];
    Scalar gamma;
    if (use_lambda)
        {
        gamma = lambda*d_diameter[idx];
        }
    else
        {
        unsigned int typ = __scalar_as_int(postype.w);
        gamma = s_gammas[typ];
        }

    // get the flow field at the current position
  //  const Scalar3 flow_vel = flow_field(make_scalar3(postype.x, postype.y, postype.z));

    // compute the random force
    Scalar coeff = fast::sqrt(Scalar(6.0) * gamma * T / dt);
    if (noiseless)
        coeff = Scalar(0.0);
    hoomd::RandomGenerator rng(RNGIdentifier::TwoStepSLLODLangevinFlow, seed, d_tag[idx], timestep);
    hoomd::UniformDistribution<Scalar> uniform(-coeff, coeff);
    const Scalar3 random = make_scalar3(uniform(rng), uniform(rng), uniform(rng));

    const Scalar4 velmass = d_vel[idx];
    Scalar3 vel = make_scalar3(velmass.x, velmass.y, velmass.z);
    const Scalar mass = velmass.w;

    // position
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);


    // remove flow field
    vel.x -= shear_rate*pos.y;

    // total BD force
    Scalar3 bd_force = random - gamma * (vel);

    // compute the new acceleration
    const Scalar4 net_force = d_net_force[idx];
    Scalar3 accel = make_scalar3(net_force.x,net_force.y,net_force.z);
    accel += bd_force;
    const Scalar minv = Scalar(1.0) / mass;
    accel.x *= minv;
    accel.y *= minv;
    accel.z *= minv;

    // apply sllod velocity correction
    vel.x -= Scalar(0.5)*shear_rate*vel.y*dt;

    // add flow field
    vel.x += shear_rate*pos.y;

    // update the velocity
    vel += Scalar(0.5) * dt * accel;

    // write out update velocity and acceleration
    d_vel[idx] = make_scalar4(vel.x, vel.y, vel.z, mass);
    d_accel[idx] = accel;
    }



} // end namespace kernel

hipError_t langevin_sllod_step1(Scalar4 *d_pos,
                                int3 *d_image,
                                Scalar4 *d_vel,
                                const Scalar3 *d_accel,
                                const unsigned int *d_group,
                                const BoxDim& box,
                                const unsigned int N,
                                const Scalar dt,
                                const Scalar shear_rate,
                                const bool flipped,
                                const Scalar boundary_shear_velocity,
                                const unsigned int block_size)
    {
    if (N == 0) return hipSuccess;

    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))kernel::langevin_sllod_step1);
        max_block_size = attr.maxThreadsPerBlock;
        }

    const int run_block_size = min(block_size, max_block_size);
    kernel::langevin_sllod_step1<<<N/run_block_size+1, run_block_size>>>(d_pos,
                                                                        d_image,
                                                                        d_vel,
                                                                        d_accel,
                                                                        d_group,
                                                                        box,
                                                                        N,
                                                                        dt,
                                                                        shear_rate,
                                                                        flipped,
                                                                        boundary_shear_velocity);
    return hipSuccess;
    }

hipError_t langevin_sllod_step2(Scalar4 *d_vel,
                                Scalar3 *d_accel,
                                const Scalar4 *d_pos,
                                const Scalar4 *d_net_force,
                                const unsigned int *d_tag,
                                const unsigned int *d_group,
                                const Scalar *d_diameter,
                                const Scalar lambda,
                                const Scalar *d_gamma,
                                const unsigned int ntypes,
                                const unsigned int N,
                                const Scalar dt,
                                const Scalar T,
                                const unsigned int timestep,
                                const unsigned int seed,
                                bool noiseless,
                                bool use_lambda,
                                const Scalar shear_rate,
                                const unsigned int block_size)
    {
    if (N == 0) return hipSuccess;

    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))kernel::langevin_sllod_step2);
        max_block_size = attr.maxThreadsPerBlock;
        }

    const int run_block_size = min(block_size, max_block_size);
    const size_t shared_bytes = sizeof(Scalar) * ntypes;

    kernel::langevin_sllod_step2<<<N/run_block_size+1, run_block_size, shared_bytes>>>(d_vel,
                                                               d_accel,
                                                               d_pos,
                                                               d_net_force,
                                                               d_tag,
                                                               d_group,
                                                               d_diameter,
                                                               lambda,
                                                               d_gamma,
                                                               ntypes,
                                                               N,
                                                               dt,
                                                               T,
                                                               timestep,
                                                               seed,
                                                               noiseless,
                                                               use_lambda,
                                                               shear_rate);
    return hipSuccess;
    }



} // end namespace gpu
} // end namespace azplugins
