#include "hip/hip_runtime.h"
// Copyright (c) 2018-2020, Michael P. Howard
// This file is part of the azplugins project, released under the Modified BSD License.

// Maintainer: astatt

/*!
 * \file TwoStepSLLODNVTFlowGPU.cu
 * \brief Declaration of SLLOD equation of motion with NVT Nosé-Hoover thermostat
 */

#include "TwoStepSLLODNVTFlowGPU.cuh"
#include <assert.h>

namespace azplugins
{
namespace gpu
{
namespace kernel
{
/*! \file TwoStepNVTGPU.cu
    \brief Defines GPU kernel code for NVT integration on the GPU. Used by TwoStepNVTGPU.
*/

//! Takes the first 1/2 step forward in the NVT integration step
/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_image array of particle images
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param work_size Number of members in the group for this GPU
    \param box Box dimensions for periodic boundary condition handling
    \param exp_fac Velocity rescaling factor from thermostat
    \param deltaT Amount of real time to step forward in one time step
    \param shear_rate Shear rate of box deformation
    \param flipped True if the box is flipped this time step
    \param boundary_shear_velocity Shear velocity at the pbc boundary
    \param offset The offset of this GPU into the list of particles

    Take the first half step forward in the NVT integration.

    See gpu_nve_step_one_kernel() for some performance notes on how to handle the group data reads efficiently.
*/
extern "C" __global__
void sllod_nvt_step_one(Scalar4 *d_pos,
                             Scalar4 *d_vel,
                             const Scalar3 *d_accel,
                             int3 *d_image,
                             unsigned int *d_group_members,
                             unsigned int work_size,
                             BoxDim box,
                             Scalar exp_fac,
                             Scalar deltaT,
                             Scalar shear_rate,
                             bool flipped,
                             Scalar boundary_shear_velocity,
                             unsigned int offset)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < work_size)
        {
        unsigned int idx = d_group_members[group_idx + offset];

        // update positions to the next timestep and update velocities to the next half step
        Scalar4 postype = d_pos[idx];
        Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);

        Scalar4 velmass = d_vel[idx];
        Scalar3 vel = make_scalar3(velmass.x, velmass.y, velmass.z);
        Scalar3 accel = d_accel[idx];


        // remove flow field
        vel.x -= shear_rate*pos.y;

        // rescale velocity
        vel *= exp_fac;

        // apply sllod velocity correction
        vel.x -= Scalar(0.5)*shear_rate*vel.y*deltaT;

        // add flow field
        vel.x += shear_rate*pos.y;

        // update velocity
        vel += Scalar(0.5)*accel*deltaT;

        // update position
        pos += deltaT * vel;


        // read in the image flags
        int3 image = d_image[idx];

        // if box deformation caused a flip, wrap pos back into box
        if (flipped){
          d_image[idx].x += d_image[idx].y;
        //    pos.x *= -1;
        }

        // time to fix the periodic boundary conditions
        box.wrap(pos, image);

        // Periodic boundary correction to velocity:
        // if particle leaves from (+/-) y boundary it gets (-/+) velocity at boundary
        // note carefully that pair potentials dependent on differences in
        // velocities (e.g. DPD) are not yet explicitly supported.

        if ((image.y-d_image[idx].y)==1) // crossed pbc in +y, image increased by 1
        {
          vel.x -= boundary_shear_velocity;
        }
        else if ((image.y-d_image[idx].y)==-1) // crossed pbc in -y, image decreased by 1
        {
          vel.x += boundary_shear_velocity;
        }

        // write out the results
        d_pos[idx] = make_scalar4(pos.x, pos.y, pos.z, postype.w);
        d_vel[idx] = make_scalar4(vel.x, vel.y, vel.z, velmass.w);
        d_image[idx] = image;
        }
    }

//! Takes the second 1/2 step forward in the NVT integration step
/*! \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param work_size Number of members in the group for this GPU
    \param d_net_force Net force on each particle
    \param deltaT Amount of real time to step forward in one time step
    \param offset The offset of this GPU into the list of particles
*/
extern "C" __global__
void sllod_nvt_step_two(Scalar4 *d_vel,
                             Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int work_size,
                             Scalar4 *d_net_force,
                             Scalar deltaT,
                             Scalar shear_rate,
                             Scalar exp_v_fac_thermo,
                             unsigned int offset)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < work_size)
        {
        unsigned int idx = d_group_members[group_idx+offset];

        // read in the net force and calculate the acceleration
        Scalar4 net_force = d_net_force[idx];
        Scalar3 accel = make_scalar3(net_force.x,net_force.y,net_force.z);

        Scalar4 vel = d_vel[idx];
        Scalar3 v = make_scalar3(vel.x,vel.y,vel.z);

        Scalar mass = vel.w;
        accel = accel/mass;

        // rescale
        v *= exp_v_fac_thermo;

        // SLLOD correction to velocity: shear rate tensor dotted with velocity
        const Scalar3 v_del_u = make_scalar3(shear_rate* vel.y, 0.0, 0.0);

        // update velocity
        v += Scalar(0.5)*(accel - v_del_u)*deltaT;

        // write out data
        d_vel[idx] = make_scalar4(v.x,v.y,v.z,vel.w);

        // since we calculate the acceleration, we need to write it for the next step
        d_accel[idx] = accel;
        }
    }




} //end namespace kernel

/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_image array of particle images
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param box Box dimensions for periodic boundary condition handling
    \param block_size Size of the block to run
    \param exp_fac Thermostat rescaling factor
    \param deltaT Amount of real time to step forward in one time step
    \param flipped true if box is flipped in this timestep
    \param boundary_shear_velocity value of the shear velocity at pbc boundary
*/
hipError_t sllod_nvt_step_one(Scalar4 *d_pos,
                             Scalar4 *d_vel,
                             const Scalar3 *d_accel,
                             int3 *d_image,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             const BoxDim& box,
                             unsigned int block_size,
                             Scalar exp_fac,
                             Scalar deltaT,
                             Scalar shear_rate,
                             bool flipped,
                             Scalar boundary_shear_velocity,
                             const GPUPartition& gpu_partition)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))kernel::sllod_nvt_step_one);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // setup the grid to run the kernel
        dim3 grid( (nwork/run_block_size) + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        // run the kernel, starting with offset range.first
        kernel::sllod_nvt_step_one<<< grid, threads >>>(d_pos,
                             d_vel,
                             d_accel,
                             d_image,
                             d_group_members,
                             nwork,
                             box,
                             exp_fac,
                             deltaT,
                             shear_rate,
                             flipped,
                             boundary_shear_velocity,
                             range.first);
        }

    return hipSuccess;
    }


/*! \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_group_members Device array listing the indices of the members of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Net force on each particle
    \param block_size Size of the block to execute on the device
    \param deltaT Amount of real time to step forward in one time step
    \param shear_rate Box deformation shear rate
    \param exp_v_fac_thermo Exponential velocity scaling factor
*/
hipError_t sllod_nvt_step_two(Scalar4 *d_vel,
                             Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             Scalar4 *d_net_force,
                             unsigned int block_size,
                             Scalar deltaT,
                             Scalar shear_rate,
                             Scalar exp_v_fac_thermo,
                             const GPUPartition& gpu_partition)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))kernel::sllod_nvt_step_two);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);

    // iterate over active GPUs in reverse, to end up on first GPU when returning from this function
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;

        // setup the grid to run the kernel
        dim3 grid( (nwork/run_block_size) + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        // run the kernel
        kernel::sllod_nvt_step_two<<< grid, threads >>>(d_vel, d_accel, d_group_members, nwork, d_net_force, deltaT, shear_rate, exp_v_fac_thermo, range.first);
        }

    return hipSuccess;
    }

} //end namespace gpu
} //end namespace azplugins
